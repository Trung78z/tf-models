#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n)
        c[i] = a[i] + b[i];
}

int main() {
    const int N = 256;
    size_t size = N * sizeof(int);

    // Cấp phát bộ nhớ trên host
    int *h_a = new int[N];
    int *h_b = new int[N];
    int *h_c = new int[N];

    // Khởi tạo dữ liệu
    for (int i = 0; i < N; ++i) {
        h_a[i] = i;
        h_b[i] = 2 * i;
    }

    // Cấp phát bộ nhớ trên device
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copy dữ liệu từ host sang device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Gọi kernel (sử dụng nhiều block nếu cần)
    int threadsPerBlock = 128;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);

    // Copy kết quả từ device về host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // In kết quả (tùy chọn)
    for (int i = 0; i < 10; ++i)
        std::cout << h_c[i] << " ";
    std::cout << std::endl;

    // Giải phóng bộ nhớ
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
